
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fill_int(int *array_to_fill, int size)
{
    for (int i = 0; i < size; ++i)
    {
        array_to_fill[i] = i;
    }
}

void print_array(int* array, int size) {
    printf("[ ");
    for (int i = 0; i < size; ++i)
    {
        printf("%d ", array[i]);
    }
    printf("]\n");
}

int main(void)
{
    int *a, *b, *c;
    int *gpu_a, *gpu_b, *gpu_c;
    int size = N * sizeof(int);
    // allocation de l’espace pour le device
    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    fill_int(a, N);
    fill_int(b, N);

    // Copie des donnees vers le Device
    hipMemcpy(gpu_a, &a, size, hipMemcpyHostToDevice);
    // checkCudaErrors(cudaMemcpy(gpu_a, &a, size, cudaMemcpyHostToDevice));
    hipMemcpy(gpu_b, &b, size, hipMemcpyHostToDevice);
    add<<<N, 1>>>(gpu_a, gpu_b, gpu_c);
    // Copie du resultat vers Host
    hipMemcpy(&c, gpu_c, size, hipMemcpyDeviceToHost);

    // print_array(c, N);

    // Liberation de l’espace alloue
    free(a);
    free(b);
    free(c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}