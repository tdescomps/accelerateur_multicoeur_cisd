
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main(void)
{
    int a, b, c;
    int *gpu_a, *gpu_b, *gpu_c;
    int size = sizeof(int);
    // allocation de l’espace pour le device
    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);
    a = 2;
    b = 7;

    // Copie des donnees vers le Device
    hipMemcpy(gpu_a, &a, size, hipMemcpyHostToDevice);
    // checkCudaErrors(cudaMemcpy(gpu_a, &a, size, cudaMemcpyHostToDevice));
    hipMemcpy(gpu_b, &b, size, hipMemcpyHostToDevice);
    add<<<1, 1>>>(gpu_a, gpu_b, gpu_c);
    // Copie du resultat vers Host
    hipMemcpy(&c, gpu_c, size, hipMemcpyDeviceToHost);
    // Liberation de l’espace alloue
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    printf("Computed on device: %d + %d = %d\n", a, b, c);

    return 0;
}