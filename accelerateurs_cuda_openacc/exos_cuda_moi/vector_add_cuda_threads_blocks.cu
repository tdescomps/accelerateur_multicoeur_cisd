
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 1000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n)
	{
		out[idx] = a[idx] + b[idx];
	}
}

int main()
{
	float *a, *b, *out;
	float *d_a, *d_b, *d_out;

	// Allocate memory
	a = (float *)malloc(sizeof(float) * N);
	b = (float *)malloc(sizeof(float) * N);
	out = (float *)malloc(sizeof(float) * N);

	// Allocate memory on GPU
	hipMalloc((void **)&d_a, sizeof(float) * N);
	hipMalloc((void **)&d_b, sizeof(float) * N);
	hipMalloc((void **)&d_out, sizeof(float) * N);

	// Initialize array
	for (int i = 0; i < N; i++)
	{
		a[i] = i * 1.0f;
		b[i] = i * 2.0f;
	}

	hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

	// Main function
	int block_size = 256;
    int grid_size = ((N + block_size - 1) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

	hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

	// Verification
	for (int i = 0; i < N; i++)
	{
		if (fabs(out[i] - a[i] - b[i]) > MAX_ERR)
		{
			printf("error\n");
			break;
		}
	}

	printf("out[2] = %f\n", out[2]);

	free(a);
	free(b);
	free(out);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

	return 0;
}
